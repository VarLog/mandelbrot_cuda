#include "hip/hip_runtime.h"
/*
 * Mandelbrot set on CUDA.
 *
 * Copyright (C) Fedorenko Maxim <varlllog@gmail.com>
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#include <math.h>

#include "utils.h"
#include "hip/hip_complex.h"
#include "mandelbrot.h"

//#define DIMX (64*4*4)
//#define DIMY (64*4*3)

#define ITER_COUNT 31

__device__ unsigned char mandelbrot( int x, int y, float angle ) {
    // project the screen coordinate into the complex plane
    const float cx = -2.0f + 3.0f * x / gridDim.x;
    const float cy = -1.125f + 2.25f * y / gridDim.y;
    
    // rotate
    const float angleCos = cosf( angle );
    const float angleSin = sinf( angle );

    const float rotationCenterX = -0.5f;
    const float rcx = rotationCenterX + angleCos * (cx - rotationCenterX) - angleSin * cy;
    const float rcy = angleCos * cy + angleSin * (cx - rotationCenterX);

    hipComplex z(0.0, 0.0);
    hipComplex c(rcx, rcy);

    // Mandelbrot set
    // z_0 = 0
    // z_n = z_(n-1)^2 + c
    // |z_n| <= 4
    unsigned char i = 0;
    for (i=0; i<ITER_COUNT; i++) {
        z = z * z + c;
        if (z.magnitude2() > 4)
            return i+1;
    }

    return 0;
}

__global__ void kernel( unsigned char *ptr, float angle ) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    unsigned char mandelbrotValue = mandelbrot( x, y, angle );
    ptr[offset] = mandelbrotValue;
}

unsigned char* calculate_mandelbrot( int w, int h, float angle ) {
    unsigned char *data_buf = (unsigned char*)calloc( w*h, sizeof(unsigned char) );
    
    unsigned char *dev_data_buf;

    HANDLE_ERROR( hipMalloc( (void**)&dev_data_buf, w*h*sizeof(unsigned char) ) );

    dim3 grid(w,h);
    kernel<<<grid,1>>>( dev_data_buf, angle );

    HANDLE_ERROR( hipMemcpy( data_buf, dev_data_buf,
                              w*h*sizeof(unsigned char),
                              hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipFree( dev_data_buf ) );
  
    return data_buf;
}

